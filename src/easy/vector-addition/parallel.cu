
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void vector_addition(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    srand(time(0));

    int N = 50000000;

    size_t size = sizeof(float) * N;

    std::vector<float> a(N);
    std::vector<float> b(N);
    std::vector<float> c(N);

    for (int i = 0; i < N; i++) {
        a[i] = (float)rand() / RAND_MAX;
        b[i] = (float)rand() / RAND_MAX;
    }

    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int numberOfBlocks = (N / threadsPerBlock) + 1;

    std::cout << "-> Running with " << numberOfBlocks << " blocks!"
              << std::endl;
    std::cout << "-> Running with " << threadsPerBlock << " threads per block!"
              << std::endl;

    hipEvent_t start, end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    float elapsed = 0.0f;

    vector_addition<<<numberOfBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipEventElapsedTime(&elapsed, start, end);

    std::cout << "-> Elapsed: " << elapsed << std::endl;

    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "==========" << std::endl;

    for (int i = 0; i < 10; i++) {
        std::cout << "-> Result: " << c[i] << std::endl;
    }

    return 0;
}