
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void vector_addition(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    srand(time(0));

    int N = 100000000;

    size_t size = sizeof(float) * N;

    std::vector<float> a(N);
    std::vector<float> b(N);
    std::vector<float> c(N);

    for (int i = 0; i < N; i++) {
        a[i] = (float)rand() / RAND_MAX;
        b[i] = (float)rand() / RAND_MAX;
    }

    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "-> Running with " << numberOfBlocks << " blocks!"
              << std::endl;
    std::cout << "-> Running with " << threadsPerBlock << " threads per block!"
              << std::endl;

    hipEvent_t start, end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    float elapsed = 0.0f;

    hipEventRecord(start, 0);

    vector_addition<<<numberOfBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipEventRecord(end, 0);

    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    std::cout << "-> Elapsed: " << elapsed << " ms" << std::endl;

    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "==========" << std::endl;
    std::cout << "Searching for differences..." << std::endl;

    float check, difference;
    for (int i = 0; i < N; i++) {
        check = a[i] + b[i];
        difference = abs(c[i] - check);
        if (difference > 0.0f) {
            std::cout << "-> Found difference!" << std::endl;
        }
    }

    return 0;
}